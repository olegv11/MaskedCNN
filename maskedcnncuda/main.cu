#include "hip/hip_runtime.h"
#include <assert.h>
#include "Cuda.hpp"
#include <stdio.h>

extern "C" {

__global__ void ReLuKernel(const float *__restrict__ x, float *__restrict__ y, float *__restrict__ delta, int num)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num; i += blockDim.x * gridDim.x)
    {
        y[i] = (x[i] > 0.0) ? x[i] : 0.0;
        delta[i] = (x[i] > 0.0) ? 1.0 : 0.0;
    }
}

__global__ void IdKernel(const float *__restrict__ x, float *__restrict__ y, float *__restrict__ delta, int num)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num; i += blockDim.x * gridDim.x)
    {
        y[i] = x[i];
        delta[i] = 1;
    }
}

__global__ void SigmoidKernel(const float *__restrict__ x, float *__restrict__ y, float *__restrict__ delta, int num)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num; i += blockDim.x * gridDim.x)
    {
        y[i] = 1.0 / (1.0 + expf(-x[i]));
        delta[i] = y[i] * (1 - y[i]);
    }
}

__global__ void TanhKernel(const float *__restrict__ x, float *__restrict__ y, float *__restrict__ delta, int num)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num; i += blockDim.x * gridDim.x)
    {
        float t = expf(2 * x[i]);
        y[i] = (t - 1) / (t + 1);
        delta[i] = (1 - y[i] * y[i]);
    }
}


void ReLu_activate_gpu(const float *__restrict__ x, float *__restrict__ y, float *__restrict__ delta, int num)
{
    ReLuKernel<<<GET_BLOCKS(num), NUM_THREADS>>>(x,y,delta,num);
}

void Id_activate_gpu(const float *__restrict__ x, float *__restrict__ y, float *__restrict__ delta, int num)
{
    IdKernel<<<GET_BLOCKS(num), NUM_THREADS>>>(x,y,delta,num);
}

void Sigmoid_activate_gpu(const float *__restrict__ x, float *__restrict__ y, float *__restrict__ delta, int num)
{
    SigmoidKernel<<<GET_BLOCKS(num), NUM_THREADS>>>(x,y,delta,num);
}

void Tanh_activate_gpu(const float *__restrict__ x, float *__restrict__ y, float *__restrict__ delta, int num)
{
    TanhKernel<<<GET_BLOCKS(num), NUM_THREADS>>>(x,y,delta,num);
}

__global__ void im2col_gpu_kernel(const int n, const float* dataIm,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int height_col, const int width_col,
    float* dataCol)
{
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < n; index += blockDim.x * gridDim.x)
    {
        const int h_index = index / width_col;
        const int h_col = h_index % height_col;
        const int w_col = index % width_col;
        const int c_im = h_index / height_col;
        const int c_col = c_im * kernel_h * kernel_w;
        const int h_offset = h_col * stride_h - pad_h;
        const int w_offset = w_col * stride_w - pad_w;
        float* data_col_ptr = dataCol;
        data_col_ptr += (c_col * height_col + h_col) * width_col + w_col;
        const float* data_im_ptr = dataIm;
        data_im_ptr += (c_im * height + h_offset) * width + w_offset;
        for (int i = 0; i < kernel_h; ++i)
        {
            for (int j = 0; j < kernel_w; ++j)
            {
                int h_im = h_offset + i;
                int w_im = w_offset + j;
                *data_col_ptr =
                    (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) ?
                    data_im_ptr[i * width + j] : 0;
                //if (index == 7)
                //printf("data_col_ptr = %f\n", (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) ?
                //           data_im_ptr[i * width + j] : 0);
                data_col_ptr += height_col * width_col;
            }
        }
    }
}



void im2col_gpu(const float *dataIm, int inputChannels, int inputHeight, int inputWidth,
                int filterSize, int pad, int stride, float *dataCol)
{
    const int outputHeight = (inputHeight + 2 * pad - filterSize) / stride + 1;
    const int outputWidth = (inputWidth + 2 * pad - filterSize) / stride + 1;
    int numKernels = inputChannels * outputHeight * outputWidth;
    //printf("HEY HEY HEY %d %d %d %d %d %d %d %d\n", numKernels, inputHeight, inputWidth, filterSize, pad, stride, outputHeight, outputWidth);
    im2col_gpu_kernel<<<(inputChannels + 512 - 1) / 512, 512>>>(numKernels, dataIm, inputHeight, inputWidth, filterSize, filterSize, pad,
                       pad, stride, stride, outputHeight, outputWidth, dataCol);
}

void hello_world()
{
    printf("ahahaha!!!\n");
}

}
